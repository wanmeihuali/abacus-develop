#include "hip/hip_runtime.h"
#include "sltk_grid.cuh"
#include "../src_pw/tools.h"
#include "sltk_atom_input.h"

void Construct_Adjacent_expand_Cuda(Grid *grid, const int true_i,
                                    const int true_j, const int true_k) {
    //	if (test_grid)TITLE(ofs_running, "Grid", "Construct_Adjacent_expand");

    //----------------------------------------------------------
    // EXPlAIN : In expand grid case, use
    // AdjacentSet::index_expand() to record the grid number,
    // We use formula (i*dy*dz + j*dz + k) to store the
    // displacement of cell.
    // Of course , an alternative operatiion is to store the
    // (i,j,k),but we want to use memory as small as possible
    // for this storage.
    //----------------------------------------------------------
    AdjacentSet::setExpandFlag(grid->expand_flag);

    AdjacentSet::setDx(grid->dx);

    AdjacentSet::setDy(grid->dy);

    AdjacentSet::setDz(grid->dz);

    // mohan add 2009-10-20
    AdjacentSet::setTrueX(true_i);

    AdjacentSet::setTrueY(true_j);

    AdjacentSet::setTrueZ(true_k);

    AdjacentSet::setCenter(true_i * grid->dy * grid->dz + true_j * grid->dz +
                           true_k);

    //	if(test_grid)OUT(ofs_running,"GridCenter",true_i,true_j,true_k);
    //	if(test_grid)OUT(ofs_running,"GridDim",dx,dy,dz);

    //-----------------------------------------------------------
    // EXPLAIN : (true_i,true_j,true_k) is the cell we want
    // to found AdjacentSet.And other cell save the displacement
    // of center_grid in 'in_grid'
    //-----------------------------------------------------------
    for (int i = 0; i < grid->dx; i++) {
        for (int j = 0; j < grid->dy; j++) {
            for (int k = 0; k < grid->dz; k++) {
                grid->Cell[i][j][k].in_grid[0] = i - true_i;
                grid->Cell[i][j][k].in_grid[1] = j - true_j;
                grid->Cell[i][j][k].in_grid[2] = k - true_k;
            }
        }
    }

    //----------------------------------------------------------
    // EXPLAIN : Only construct AdjacentSet for 'true' cell.
    //----------------------------------------------------------
    // for (int ia = 0;ia < Cell[true_i][true_j][true_k].length;ia++)
    int true_ia_range = grid->Cell[true_i][true_j][true_k].length;
    for (int true_ia = 0; true_ia < true_ia_range; ++true_ia) {
        grid->Cell[true_i][true_j][true_k]
            .address[true_ia]
            .fatom.allocate_AdjacentSet();
    }

    if (grid->pbc) {
        auto& selected_cell = grid->Cell[true_i][true_j][true_k];
        for (int true_ia = 0; true_ia < true_ia_range; ++true_ia) {
            for (int i = 0; i < grid->dx; i++) {
                for (int j = 0; j < grid->dy; j++) {
                    for (int k = 0; k < grid->dz; k++) {
                        for (int ia = 0; ia < grid->Cell[i][j][k].length;
                             ia++) {
                            const double x = selected_cell
                                                 .address[true_ia]
                                                 .fatom.x();
                            const double y = selected_cell
                                                 .address[true_ia]
                                                 .fatom.y();
                            const double z = selected_cell
                                                 .address[true_ia]
                                                 .fatom.z();

                            double x2 =
                                grid->Cell[i][j][k].address[ia].fatom.x();
                            double y2 =
                                grid->Cell[i][j][k].address[ia].fatom.y();
                            double z2 =
                                grid->Cell[i][j][k].address[ia].fatom.z();
                            const int b0 = grid->Cell[i][j][k].in_grid[0];
                            const int b1 = grid->Cell[i][j][k].in_grid[1];
                            const int b2 = grid->Cell[i][j][k].in_grid[2];

                            if (!grid->expand_flag) {
                                x2 = x2 + b0 * grid->vec1[0] +
                                     b1 * grid->vec2[0] + b2 * grid->vec3[0];
                                y2 = y2 + b0 * grid->vec1[1] +
                                     b1 * grid->vec2[1] + b2 * grid->vec3[1];
                                z2 = z2 + b0 * grid->vec1[2] +
                                     b1 * grid->vec2[2] + b2 * grid->vec3[2];
                            }
                            double delta_x = x - x2;
                            double delta_y = y - y2;
                            double delta_z = z - z2;

                            double dr =
                                sqrt(delta_x * delta_x + delta_y * delta_y +
                                     delta_z * delta_z);

                            if (dr != 0.0 && dr <= grid->sradius) {
                                int offset = grid->Cell[i][j][k].address -
                                             grid->atomlink;
                                offset += ia;
                                selected_cell
                                    .address[true_ia]
                                    .fatom.getAdjacentSet()
                                    ->set(b0, b1, b2, offset, test_grid);
                            }
                        }
                    }
                }
            }
        }
    } else {
        WARNING_QUIT("Construct_Adjacent_expand",
                     "\n Expand case, must use periodic boundary.");
    }
    return;
}
