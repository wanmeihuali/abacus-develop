#include "hip/hip_runtime.h"
#include "sltk_grid.cuh"

#include <cstdint>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <vector>
#include <iostream>
#include <boost/progress.hpp>

#include "../src_pw/tools.h"
#include "sltk_atom_input.h"

__global__ void Construct_adjacent_expand_kernel(
    uint64_t const dx,
    uint64_t const dy,
    uint64_t const dz,
    uint64_t const* const cell_offset,
    double const* const atom_link_x,
    double const* const atom_link_y,
    double const* const atom_link_z,
    double const* const cell_b0,
    double const* const cell_b1,
    double const* const cell_b2,
    double const* const grid_vec3x3
) {

}

void Construct_Adjacent_expand_Cuda(Grid *grid, const int true_i,
                                    const int true_j, const int true_k) {

    if (!grid->pbc) {
        WARNING_QUIT("Construct_Adjacent_expand",
                     "\n Expand case, must use periodic boundary.");
        return;
    }
    //	if (test_grid)TITLE(ofs_running, "Grid", "Construct_Adjacent_expand");

    //----------------------------------------------------------
    // EXPlAIN : In expand grid case, use
    // AdjacentSet::index_expand() to record the grid number,
    // We use formula (i*dy*dz + j*dz + k) to store the
    // displacement of cell.
    // Of course , an alternative operatiion is to store the
    // (i,j,k),but we want to use memory as small as possible
    // for this storage.
    //----------------------------------------------------------
    AdjacentSet::setExpandFlag(grid->expand_flag);

    AdjacentSet::setDx(grid->dx);

    AdjacentSet::setDy(grid->dy);

    AdjacentSet::setDz(grid->dz);

    // mohan add 2009-10-20
    AdjacentSet::setTrueX(true_i);

    AdjacentSet::setTrueY(true_j);

    AdjacentSet::setTrueZ(true_k);

    AdjacentSet::setCenter(true_i * grid->dy * grid->dz + true_j * grid->dz +
                           true_k);

    //	if(test_grid)OUT(ofs_running,"GridCenter",true_i,true_j,true_k);
    //	if(test_grid)OUT(ofs_running,"GridDim",dx,dy,dz);

    //-----------------------------------------------------------
    // EXPLAIN : (true_i,true_j,true_k) is the cell we want
    // to found AdjacentSet.And other cell save the displacement
    // of center_grid in 'in_grid'
    //-----------------------------------------------------------
    for (int i = 0; i < grid->dx; i++) {
        for (int j = 0; j < grid->dy; j++) {
            for (int k = 0; k < grid->dz; k++) {
                grid->Cell[i][j][k].in_grid[0] = i - true_i;
                grid->Cell[i][j][k].in_grid[1] = j - true_j;
                grid->Cell[i][j][k].in_grid[2] = k - true_k;
            }
        }
    }

    //----------------------------------------------------------
    // EXPLAIN : Only construct AdjacentSet for 'true' cell.
    //----------------------------------------------------------
    // for (int ia = 0;ia < Cell[true_i][true_j][true_k].length;ia++)
    int true_ia_range = grid->Cell[true_i][true_j][true_k].length;
    for (int true_ia = 0; true_ia < true_ia_range; ++true_ia) {
        grid->Cell[true_i][true_j][true_k]
            .address[true_ia]
            .fatom.allocate_AdjacentSet();
    }

    uint64_t const cell_count{grid->dx * grid->dy * grid->dz};
    thrust::host_vector<uint64_t> cell_offset(cell_count + 1);
    thrust::host_vector<double> atom_link_x;
    thrust::host_vector<double> atom_link_y;
    thrust::host_vector<double> atom_link_z;

    thrust::host_vector<double> cell_b0(cell_count);
    thrust::host_vector<double> cell_b1(cell_count);
    thrust::host_vector<double> cell_b2(cell_count);
    {
        std::cout << "Preparing data..." << std::endl;
        boost::progress_display show_progress(cell_count);
        uint64_t current_offset{0U};
        uint64_t cell_idx{0U};
        for (int i{0}; i < grid->dx; i++) {
            for (int j{0}; j < grid->dy; j++) {
                for (int k{0}; k < grid->dz; k++) {
                    cell_offset[cell_idx] = current_offset;
                    cell_b0[cell_idx] = grid->Cell[i][j][k].in_grid[0];
                    cell_b1[cell_idx] = grid->Cell[i][j][k].in_grid[1];
                    cell_b2[cell_idx] = grid->Cell[i][j][k].in_grid[2];
                    ++cell_idx;
                    ++show_progress;
                    current_offset += grid->Cell[i][j][k].length;
                    for (int64_t ia{0}; ia < grid->Cell[i][j][k].length; ++ia) {
                        atom_link_x.push_back(grid->Cell[i][j][k].address[ia].fatom.x());
                        atom_link_y.push_back(grid->Cell[i][j][k].address[ia].fatom.y());
                        atom_link_z.push_back(grid->Cell[i][j][k].address[ia].fatom.z());
                    }
                }
            }
        }   
        cell_offset[cell_idx] = current_offset;
    }

    std::cout << "Copying data to GPU..." << std::endl;
    thrust::device_vector<uint64_t> cell_offset_device = cell_offset;
    thrust::device_vector<double> atom_link_x_device = atom_link_x;
    thrust::device_vector<double> atom_link_y_device = atom_link_y;
    thrust::device_vector<double> atom_link_z_device = atom_link_z;
    thrust::device_vector<double> cell_b0_device = cell_b0;
    thrust::device_vector<double> cell_b1_device = cell_b1;
    thrust::device_vector<double> cell_b2_device = cell_b2;
    thrust::device_vector<double> grid_vec(9);
    {
        auto raw_ptr = thrust::raw_pointer_cast(grid_vec.data());
        hipMemcpy(raw_ptr, grid->vec1, sizeof(double) * 3, hipMemcpyHostToDevice);
    }
    thrust::device_vector<double> grid_vec2(3);
    {
        auto raw_ptr = thrust::raw_pointer_cast(grid_vec.data()) + 3;
        hipMemcpy(raw_ptr, grid->vec2, sizeof(double) * 3, hipMemcpyHostToDevice);
    }
    thrust::device_vector<double> grid_vec3(3);
    {
        auto raw_ptr = thrust::raw_pointer_cast(grid_vec.data()) + 6;
        hipMemcpy(raw_ptr, grid->vec3, sizeof(double) * 3, hipMemcpyHostToDevice);
    }
    std::cout << "Start computing..." << std::endl;
    dim3 cuda_grid{true_ia_range, cell_count};
    dim3 cuda_block{1024};
    Construct_adjacent_expand_kernel<<<cuda_grid, cuda_block>>>(
        grid->dx,
        grid->dy,
        grid->dz,
        thrust::raw_pointer_cast(cell_offset.data()),
        thrust::raw_pointer_cast(atom_link_x.data()),
        thrust::raw_pointer_cast(atom_link_y.data()),
        thrust::raw_pointer_cast(atom_link_z.data()),
        thrust::raw_pointer_cast(cell_b0.data()),
        thrust::raw_pointer_cast(cell_b1.data()),
        thrust::raw_pointer_cast(cell_b2.data()),
        thrust::raw_pointer_cast(grid_vec.data())
    );
    // TODO: make it parallel in CUDA.
    auto& selected_cell = grid->Cell[true_i][true_j][true_k];
    for (int true_ia = 0; true_ia < true_ia_range; ++true_ia) {
        for (int i = 0; i < grid->dx; i++) {
            for (int j = 0; j < grid->dy; j++) {
                for (int k = 0; k < grid->dz; k++) {
                    for (int ia = 0; ia < grid->Cell[i][j][k].length;
                            ia++) {
                        const double x = selected_cell
                                                .address[true_ia]
                                                .fatom.x();
                        const double y = selected_cell
                                                .address[true_ia]
                                                .fatom.y();
                        const double z = selected_cell
                                                .address[true_ia]
                                                .fatom.z();

                        double x2 =
                            grid->Cell[i][j][k].address[ia].fatom.x();
                        double y2 =
                            grid->Cell[i][j][k].address[ia].fatom.y();
                        double z2 =
                            grid->Cell[i][j][k].address[ia].fatom.z();
                        const int b0 = grid->Cell[i][j][k].in_grid[0];
                        const int b1 = grid->Cell[i][j][k].in_grid[1];
                        const int b2 = grid->Cell[i][j][k].in_grid[2];

                        if (!grid->expand_flag) {
                            x2 = x2 + b0 * grid->vec1[0] +
                                    b1 * grid->vec2[0] + b2 * grid->vec3[0];
                            y2 = y2 + b0 * grid->vec1[1] +
                                    b1 * grid->vec2[1] + b2 * grid->vec3[1];
                            z2 = z2 + b0 * grid->vec1[2] +
                                    b1 * grid->vec2[2] + b2 * grid->vec3[2];
                        }
                        double delta_x = x - x2;
                        double delta_y = y - y2;
                        double delta_z = z - z2;

                        double dr =
                            sqrt(delta_x * delta_x + delta_y * delta_y +
                                    delta_z * delta_z);

                        if (dr != 0.0 && dr <= grid->sradius) {
                            int offset = grid->Cell[i][j][k].address -
                                            grid->atomlink;
                            offset += ia;
                            selected_cell
                                .address[true_ia]
                                .fatom.getAdjacentSet()
                                ->set(b0, b1, b2, offset, test_grid);
                        }
                    }
                }
            }
        }
    }

    return;
}
